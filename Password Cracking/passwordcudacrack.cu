#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>



__device__ int is_a_match(char *attempt) {
char plain_password1[] = "SA9860";
char plain_password2[] = "MU1995";
char plain_password3[] = "JJ2100";
char plain_password4[] = "SP0023";
char *s = attempt;
char *p = attempt;
char *m = attempt;
char *d = attempt;
char *d1 = plain_password1;
char *d2 = plain_password2;
char *d3 = plain_password3;
char *d4 = plain_password4;
	while(*s == *d1) {
	 if(*s == '\0')
	   {
	      printf("password: %s\n",plain_password1);
	      break;
	   }
           s++;
	   d1++;
	}
  
	while(*p == *d2) {
	  if(*p == '\0')
	    {
	      printf("password: %s\n",plain_password2);
	       break;
	    }
	    p++;
	    d2++;
	  }

        while(*m == *d3) {
          if(*m == '\0')
             {
               printf("password: %s\n",plain_password3);
                break;
             }
          m++;
          d3++;
           }
       while(*d == *d4) {
          if(*d == '\0')
             {
              printf("password: %s\n",plain_password4);
              return 1;
             }
             d++;
             d4++;
            }
          return 0;
}
__global__ void kernel() {
char i1,i2,i3,i4;//variables
char password[7];
password[6] = '\0';
//block id threrad id initilizedthreadidx
int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstmatch = i;
char secondmatch = j;
password[0] = firstmatch;
password[1] = secondmatch;


		for(i1='0'; i1<='9'; i1++){
	           for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
                          for(i4='0'; i4<='9'; i4++){
				password[2] = i1;
				password[3] = i2;
				password[4] = i3;
				password[5] = i4;
			    if(is_a_match(password)) {
				}
			    else {
				//printf("tried: %s\n", password);
				}


				}
			}
		}
	}
}
//time difference
	int time_difference(struct timespec *start,
	struct timespec *finish,
	long long int *difference) {
	long long int ds = finish->tv_sec - start->tv_sec;
	long long int dn = finish->tv_nsec - start->tv_nsec;
	if(dn < 0 ) {
	ds--;
	dn += 1000000000;
	}
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}
int main() {
	struct timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);
	//kernal function that determine block and threads to use
	kernel <<<26,26>>>();
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
	return 0;
}
